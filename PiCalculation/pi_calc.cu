#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define TRIALS_PER_THREAD 1024
#define BLOCKS 256
#define THREADS 256


__global__ void gpuPiCalculate(float *estimate, hiprandState *states) {
	unsigned long id = threadIdx.x + blockDim.x * blockIdx.x;
	int pointsInCircle = 0;
	float x, y;

	hiprand_init(id, id, 0, &states[id]);  //initialize hiprand

	for (int i = 0; i < TRIALS_PER_THREAD; i++) {
		x = hiprand_uniform(&states[id]);
		y = hiprand_uniform(&states[id]);
		pointsInCircle += (x*x + y * y <= 1.0f); 
	}
	estimate[id] = 4.0f * pointsInCircle / (float)TRIALS_PER_THREAD;
}

float cpuPiCalculate(long trials) {
	float x, y;
	long pointsInCircle = 0;
	for (long i = 0; i < trials; i++) {
		x = rand() / (float)RAND_MAX;
		y = rand() / (float)RAND_MAX;
		pointsInCircle += (x * x + y * y <= 1.0f);
	}
	return 4.0f * pointsInCircle / trials;
}

int main(int argc, char *argv[]) {
	clock_t start, stop;
	float host[BLOCKS * THREADS];
	float *dev;
	hiprandState *devStates;

	start = clock();
	hipMalloc((void **)&dev, BLOCKS * THREADS * sizeof(float)); 
	hipMalloc((void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState));

	gpuPiCalculate << <BLOCKS, THREADS >> > (dev, devStates);

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost);
	float gpuPI = 0;
	for (int i = 0; i < BLOCKS * THREADS; i++) {
		gpuPI += host[i];
	}
	gpuPI /= (BLOCKS * THREADS);
	stop = clock();
	printf("GPU PI= %f\n", gpuPI);
	printf("GPU PI calculate time %f s.\n", (stop - start) / (float)CLOCKS_PER_SEC);
	
	start = clock();
	float cpuPI = cpuPiCalculate(BLOCKS * THREADS * TRIALS_PER_THREAD);
	stop = clock();
	printf("CPU PI= %f\n", cpuPI);
	printf("CPU PI calculate time %f s.\n", (stop - start) / (float)CLOCKS_PER_SEC);

	return 0;
}
